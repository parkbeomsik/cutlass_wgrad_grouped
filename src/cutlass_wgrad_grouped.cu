#include <vector>
// #include "base_operation.h"
// #include "cutlass_error.h"
#include "cutlass_wgrad_grouped.h"
#include "initialize_all.h"
#include "wgrad_grouped_operation.h"

#include "hip/hip_runtime.h"
#include "cuda_error_helper.h"

#include "cutlass/conv/conv2d_problem_size.h"


namespace cutlass_wgrad_grouped {

void * _device_problems;

std::vector<Operation *> operations;

std::vector<void *> device_workspaces;
// std::vector<void *> host_workspaces;

std::vector<OperationWithWorkspace> operations_with_workspaces;

void initialize() {

    initialize_all(operations);
}

void initialize_problems(std::vector<Conv2dConfig> const & host_configs) {

    using namespace cutlass::conv;
    
    int problem_count = host_configs.size();

    // Set problem sizes in host memory first
    std::vector<Conv2dProblemSize> host_problems;
    for (int i = 0; i < problem_count; ++i) {
        Conv2dConfig host_config = host_configs.at(i);

        // Set single problem in host
        Conv2dProblemSize problem(host_config.N, 
                                  host_config.H, host_config.W, host_config.C,
                                  host_config.K, host_config.R, host_config.S,
                                  host_config.P, host_config.Q,
                                  host_config.pad_h, host_config.pad_w,
                                  host_config.stride_h, host_config.stride_w,
                                  host_config.dilation_h, host_config.dilation_w,
                                  Mode::kCrossCorrelation);

        host_problems.push_back(problem);
    }

    assert(host_problems.size() == problem_count);

    // Set problems in device memory
    checkCudaErrors(hipMalloc(&_device_problems, 
                                (size_t)sizeof(Conv2dProblemSize)*problem_count));
    checkCudaErrors(hipMemcpy((void *)_device_problems, (void *)host_problems.data(), 
                               (size_t)sizeof(Conv2dProblemSize)*problem_count,
                               hipMemcpyHostToDevice));

    // Set problems of operations (will set tensor data ptrs later)
    wGradGroupedConfig wgrad_config = {(Conv2dProblemSize *)_device_problems, problem_count, NULL, NULL, NULL, NULL};
    for (auto operation : operations) {
        void * host_workspace = malloc(operation->get_host_workspace_size());
        operation->initialize(&wgrad_config, host_workspace);
        operations_with_workspaces.push_back(OperationWithWorkspace({operation, host_workspace}));
    }

    assert(operations_with_workspaces.size() == operations.size());

}

void finalize() {
    for (auto device_workspace : device_workspaces){
        checkCudaErrors(hipFree(device_workspace));
    }
    for (auto operation_with_workspace : operations_with_workspaces) {
        free(operation_with_workspace.host_workspace);
        free(operation_with_workspace.operation);
    }
    
    device_workspaces.clear();
    operations_with_workspaces.clear();
    operations.clear();
}

OperationWithWorkspace get_best_operation(void ** ptr_A,
                              void ** ptr_B,
                              void ** ptr_C,
                              void ** ptr_D) {

    assert(operations_with_workspaces.size() == operations.size());

    std::vector<float> runtime_ms_list;
    runtime_ms_list.resize(operations.size());

    for (int i = 0; i < operations.size(); ++i) {
        auto operation = operations.at(i);
        auto host_workspace = operations_with_workspaces.at(i).host_workspace;

        operation->update_ptrs(ptr_A, ptr_B, ptr_C, ptr_D, host_workspace);

        // Warm up
        for (int iter = 0; iter < 3; ++iter) { 
            operation->run(host_workspace);
        }

        checkCudaErrors(hipDeviceSynchronize());

        // Measure runtime

        hipEvent_t events[2];

        for (auto & event : events) {
            checkCudaErrors(hipEventCreate(&event));
        }

        // Record an event at the start of a series of GEMM operations
        checkCudaErrors(hipEventRecord(events[0]));

        Status result;
        for (int iter = 0; iter < 20; ++iter) {
            result = operation->run(host_workspace);
        }

        checkCudaErrors(hipEventRecord(events[1]));
        checkCudaErrors(hipEventSynchronize(events[1]));

        float runtime_ms;
        checkCudaErrors(hipEventElapsedTime(&runtime_ms, events[0], events[1]));

        if (result == Status::kSuccess) {
            runtime_ms_list.at(i) = runtime_ms;
        } else {
            runtime_ms_list.at(i) = 100000.0;
        }
        // 
    }

    assert(runtime_ms_list.size() == operations.size());

    float min_runtime_ms = 10000.0;
    OperationWithWorkspace best_operation {NULL, NULL};
    for (int i = 0; i < runtime_ms_list.size(); ++i) {
        if (runtime_ms_list.at(i) < min_runtime_ms) {
            min_runtime_ms = runtime_ms_list.at(i);
            best_operation = operations_with_workspaces.at(i);
        }
    }

    return best_operation;
}

Status run(OperationWithWorkspace operation_with_workspace) {
    return operation_with_workspace.operation->run(operation_with_workspace.host_workspace);
}

Status update_ptrs(OperationWithWorkspace operation_with_workspace,
                   void ** ptr_A,
                   void ** ptr_B,
                   void ** ptr_C,
                   void ** ptr_D,
                   int problem_count) {
    return operation_with_workspace.operation->update_ptrs(ptr_A, ptr_B, ptr_C, ptr_D, problem_count, operation_with_workspace.host_workspace);
}


} // namespace cutlass_wgrad_grouped